#include <stdio.h>
#include <string.h>
// #include <mpi.h>
//#include "adder.cuh"
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include "symbfact.cuh"
//#include "symbfact_reordered.cuh"
#include "wtime.h"
#include <hip/hip_runtime.h>
#include <fstream> 
int count=0;
int main(int argc, char *argv[])
{
    char message[20];
    int myrank, tag=99;
    cout<<"Number of GPUs used: "<<argv[4]<<endl;
    unsigned int global_fill_in=0;
    double max_time=0;
    double min_time=0;
    double begin_time=wtime();

    //  MPI_Status status;
    //  MPI_Init(&argc, &argv);
    //  MPI_Comm_rank(MPI_COMM_WORLD, &myrank);

    ull_t fill_count=0;
    double indiv_time=0;
    myrank=0;
    symbfact_min_id(argc,argv,myrank,fill_count,indiv_time);
   
    // MPI_Barrier(MPI_COMM_WORLD);
    // MPI_Reduce(&fill_count, &global_fill_in, 1, MPI_UNSIGNED_LONG_LONG, MPI_SUM, 0, MPI_COMM_WORLD);
    // MPI_Reduce(&indiv_time, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
    // MPI_Reduce(&indiv_time, &min_time, 1, MPI_DOUBLE, MPI_MIN, 0, MPI_COMM_WORLD);
    // MPI_Barrier(MPI_COMM_WORLD); 
    if (myrank == 0)
    {
        std::fstream timeLog;
        timeLog.open("Vary_percent_blocksi_pre2.csv",std::fstream::out | std::fstream::app);
        cout<<"Final Number of fill-ins detected: "<<global_fill_in<<endl;
        cout<<"Final Maximum time reported among the processes: "<<max_time<<" ms"<<endl;
        cout<<"Final Minimum time reported among the processes: "<<min_time<<" ms"<<endl;
        timeLog<<"Final vert_count;N_gpus;"<<"p0;"<<"p1"<<";"<<"#blocks_src"<<";"<<"min_time"<<";"<<"max_time"<<endl;
        timeLog<<argv[2]<<";"<<argv[4]<<";"<<argv[6]<<";"<<argv[7]<<";"<<argv[8]<<";"<<min_time<<";"<<max_time<<endl;
        timeLog.close();
    }
    //  MPI_Finalize();
    return 0;
}
